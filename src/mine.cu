#include "hip/hip_runtime.h"
#include "mine.cuh"
#include "config.cuh"
#include <hip/hip_runtime.h>

__global__ void hello_world()
{
    printf("Hello World from GPU!\n");
}


std::vector<
    std::pair<
        std::vector<std::string>, uint64_t>>
generate_frequent_itemsets_cpu(
    const std::unordered_map<std::vector<uint32_t>, std::pair<std::vector<uint64_t>, uint64_t>, VectorHash> &filtered_transactions,
    const std::unordered_map<uint64_t, uint64_t> &subtree_util,
    const std::unordered_map<uint64_t, uint64_t> &secondary_util,
    uint64_t min_utility)
{
    std::cout << "Hello World from CPU!" << std::endl;
    return {};
}

std::vector<
    std::pair<
        std::vector<std::string>, uint64_t>>
generate_frequent_itemsets_gpu(
    const std::unordered_map<std::vector<uint32_t>, std::pair<std::vector<uint64_t>, uint64_t>, VectorHash> &filtered_transactions,
    const std::unordered_map<uint64_t, uint64_t> &subtree_util,
    const std::unordered_map<uint64_t, uint64_t> &secondary_util,
    uint64_t min_utility)
{

    hello_world<<<1, 1>>>();
    hipDeviceSynchronize();

    return {};

}