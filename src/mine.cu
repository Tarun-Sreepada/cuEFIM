#include "hip/hip_runtime.h"
#include "mine.cuh"
#include "config.cuh"
#include "database.cuh"
#include <thrust/count.h>


extern __shared__ key_value shared_memory[];
__global__ void searchGPU(database *d_db, uint32_t *transaction_hits, uint32_t transactions_count,
                          uint32_t *candidates, uint32_t number_of_candidates, uint32_t candidate_size,
                          uint32_t *secondary, uint32_t secondary_size,
                          uint32_t *secondary_reference,
                          uint32_t *candidate_utility,
                          uint32_t *candidate_subtree_utility,
                          uint32_t *candidate_local_utility)
{
    uint32_t block_id = blockIdx.x + blockIdx.y * gridDim.x;
    uint32_t tid = threadIdx.x;
    if (block_id >= transactions_count || transaction_hits[block_id] == 0)
    {
        return;
    }
    transaction_hits[block_id] = 0;

    uint32_t transaction_start = d_db->transaction_start[block_id];
    uint32_t transaction_end = d_db->transaction_end[block_id];
    uint32_t transaction_length = transaction_end - transaction_start;

    for (uint32_t i = tid; i < transaction_length * bucket_factor; i += blockDim.x)
    {
        // shared_memory[i] = d_db->item_index[transaction_start * bucket_factor + i];
        shared_memory[i].key = d_db->item_index[transaction_start * bucket_factor + i].key;
        shared_memory[i].value = d_db->item_index[transaction_start * bucket_factor + i].value;
    }   

    __syncthreads();

    for (uint32_t i = tid; i < number_of_candidates; i += blockDim.x)
    {
        uint32_t curr_cand_util = 0;
        uint32_t curr_cand_hits = 0;
        int32_t location = -1;

        for (uint32_t j = 0; j < candidate_size; j++)
        {
            uint32_t candidate = candidates[i * candidate_size + j];
            location = query_item(shared_memory, 0, transaction_length * bucket_factor, candidate);
            if (location != -1)
            {
                curr_cand_hits++;
                curr_cand_util += d_db->item_utility[location].value;
            }
        }
        if (curr_cand_hits != candidate_size)
        {
            continue;
        }
        transaction_hits[block_id] += 1;

        atomicAdd(&candidate_utility[i], curr_cand_util);

        // location -= transaction_start; // 
        uint32_t ref = secondary_reference[i];
        uint32_t secondary_index_start = secondary_size * ref;

        // collect all utilities
        for (uint32_t j = location + 1; j < transaction_end; j++)
        {
            uint32_t item = d_db->item_utility[j].key;
            if (secondary[secondary_index_start + item]) // if the item is valid secondary
            {
                curr_cand_util += d_db->item_utility[j].value;
            }
        }

        uint32_t temp = 0;

        uint32_t subtree_local_insert_location = i * secondary_size;

        for (uint32_t j = location + 1; j < transaction_end; j++)
        {
            uint32_t item = d_db->item_utility[j].key;
            if (secondary[secondary_index_start + item]) // if the item is valid secondary
            {
                atomicAdd(&candidate_local_utility[subtree_local_insert_location + item], curr_cand_util);
                atomicAdd(&candidate_subtree_utility[subtree_local_insert_location + item], curr_cand_util - temp);
                temp += d_db->item_utility[j].value;
            }
        }

    }


}


__global__ void clean_subtree_local_utility(uint32_t number_of_candidates, uint32_t *number_of_new_candidates_per_candidate, 
                                            uint32_t *subtree_utility, uint32_t *local_utility, uint32_t secondary_size, uint32_t minimum_utility)
{
    uint32_t tid = threadIdx.x;
    if (tid >= number_of_candidates)
    {
        return;
    }


    for (uint32_t i = tid * secondary_size; i < (tid + 1) * secondary_size; i++)
    {
        uint32_t item_value = i - tid * secondary_size;

        if (subtree_utility[i] >= minimum_utility)
        {
            subtree_utility[i] = item_value;
            number_of_new_candidates_per_candidate[tid + 1]++;
        }
        else
        {
            subtree_utility[i] = 0;
        }
        if (local_utility[i] >= minimum_utility)
        {
            local_utility[i] = item_value;
        }
        else
        {
            local_utility[i] = 0;
        }
    }
    return;
}

// create_new_candidates<<<1, 1>>>(thrust::raw_pointer_cast(d_candidates.data()), thrust::raw_pointer_cast(d_candidate_subtree_utility.data()), 
//                                         number_of_candidates,thrust::raw_pointer_cast(d_new_candidates.data()), 
//                                         thrust::raw_pointer_cast(d_new_secondary_reference.data()), secondary_size, candidate_size, 
//                                         thrust::raw_pointer_cast(d_number_of_new_candidates_per_candidate.data()));


__global__ void create_new_candidates(uint32_t *candidates, uint32_t *candidate_subtree_utility, uint32_t number_of_candidates,
                                      uint32_t *new_candidates, uint32_t *new_secondary_reference, uint32_t secondary_size, uint32_t candidate_size,
                                      uint32_t *number_of_new_candidates_per_candidate)
{
    uint32_t tid = threadIdx.x;
    if (tid >= number_of_candidates)
    {
        return;
    }

    if (number_of_new_candidates_per_candidate[tid] == number_of_new_candidates_per_candidate[tid + 1])
    {
        return;
    }

    uint32_t counter = candidate_size * number_of_new_candidates_per_candidate[tid];
    uint32_t refStart = number_of_new_candidates_per_candidate[tid];

    for (uint32_t i = tid * secondary_size; i < (tid + 1) * secondary_size; i++)
    {
        if (candidate_subtree_utility[i])
        {
            for (uint32_t j = tid * (candidate_size - 1); j < (tid + 1) * (candidate_size - 1); j++)
            {
                new_candidates[counter] = candidates[j];
                counter++;
            }
            new_candidates[counter] = candidate_subtree_utility[i];
            counter++;
            new_secondary_reference[refStart] = tid;
            refStart++;
        }
    }

    return;
}

void mine_patterns(params p, std::unordered_map<std::vector<uint32_t>, std::vector<uint32_t>, VectorHash> filtered_transactions,
                   std::vector<uint32_t> primary, std::vector<uint32_t> secondary,
                   std::vector<pattern> frequent_patterns, std::unordered_map<uint32_t, std::string> &intToStr)
{
    auto start = std::chrono::high_resolution_clock::now();

    uint32_t max_transaction_length = 0;
    for (const auto &transaction : filtered_transactions)
    {
        max_transaction_length = std::max(max_transaction_length, (uint32_t)transaction.first.size());
    }

    std::cout << "Max transaction length: " << max_transaction_length << std::endl;
    std::cout << "Number of transactions: " << filtered_transactions.size() << std::endl;

    secondary.push_back(0); // add 0 to the secondary list // cba to do conversions

    std::sort(secondary.begin(), secondary.end());
    std::sort(primary.begin(), primary.end());

    std::vector<uint32_t> transaction_start;
    std::vector<uint32_t> transaction_end;
    std::vector<key_value> item_utility;

    for (const auto &transaction : filtered_transactions)
    {
        transaction_start.push_back(item_utility.size());
        for (uint32_t i = 0; i < transaction.first.size(); i++)
        {
            item_utility.push_back({transaction.first[i], transaction.second[i]});
        }
        transaction_end.push_back(item_utility.size());
    }

    std::cout << "Time to convert transactions: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start).count() << "ms" << std::endl;
    start = std::chrono::high_resolution_clock::now();

    uint32_t shared_memory_requirement = max_transaction_length * sizeof(key_value) * bucket_factor; // twice as much just to be safe // tweak later
    std::cout << "Shared memory requirement: " << shared_memory_requirement * sizeof(key_value) << std::endl;
    // query the device for the maximum shared memory per block
    int device;
    hipDeviceProp_t props;
    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);
    std::cout << "Max shared memory per block: " << props.sharedMemPerBlock << std::endl;

    if (shared_memory_requirement > props.sharedMemPerBlock)
    {
        std::cerr << "Shared memory requirement exceeds the maximum shared memory per block" << std::endl;
    }
    
    std::cout << "Time to convert transactions: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start).count() << "ms" << std::endl;
    start = std::chrono::high_resolution_clock::now();

    // Create the database
    database *d_db;
    gpuErrchk(hipMalloc(&d_db, sizeof(database)));

    uint32_t *d_transaction_start;
    uint32_t *d_transaction_end;

    key_value *d_item_utility;
    key_value *d_item_index;

    gpuErrchk(hipMalloc(&d_transaction_start, transaction_start.size() * sizeof(uint32_t)));
    gpuErrchk(hipMalloc(&d_transaction_end, transaction_end.size() * sizeof(uint32_t)));
    gpuErrchk(hipMalloc(&d_item_utility, item_utility.size() * sizeof(key_value)));
    gpuErrchk(hipMalloc(&d_item_index, item_utility.size() * bucket_factor * sizeof(key_value)));

    gpuErrchk(hipMemcpy(d_transaction_start, transaction_start.data(), transaction_start.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_transaction_end, transaction_end.data(), transaction_end.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_item_utility, item_utility.data(), item_utility.size() * sizeof(key_value), hipMemcpyHostToDevice));
    gpuErrchk(hipMemset(d_item_index, 0, item_utility.size() * bucket_factor * sizeof(key_value)));

    uint32_t transactions_count = filtered_transactions.size();
    hipMemcpy(&d_db->transactions_count, &transactions_count, sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(&d_db->transaction_start, &d_transaction_start, sizeof(uint32_t *), hipMemcpyHostToDevice);
    hipMemcpy(&d_db->transaction_end, &d_transaction_end, sizeof(uint32_t *), hipMemcpyHostToDevice);
    hipMemcpy(&d_db->item_utility, &d_item_utility, sizeof(key_value *), hipMemcpyHostToDevice);
    hipMemcpy(&d_db->item_index, &d_item_index, sizeof(key_value *), hipMemcpyHostToDevice);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    std::cout << "Time to copy transactions to GPU: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start).count() << "ms" << std::endl;
    start = std::chrono::high_resolution_clock::now();

    // print_db<<<1, 1>>>(d_db);
    // gpuErrchk(hipDeviceSynchronize());
    // gpuErrchk(hipPeekAtLastError());

    // Call the kernel
    dim3 block(block_size);
    dim3 grid((transactions_count + block.x) / block.x);

    hash_transactions<<<grid, block>>>(d_db); // each thread will handle a transaction
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    std::cout << "Time to hash transactions: " << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start).count() << "ms" << std::endl;
    start = std::chrono::high_resolution_clock::now();

    uint32_t number_of_candidates = primary.size();
    uint32_t candidate_size = 1;

    // print_db_full<<<1, 1>>>(d_db);
    // gpuErrchk(hipDeviceSynchronize());
    // gpuErrchk(hipPeekAtLastError());

    thrust::device_vector<uint32_t> d_candidates = primary;
    thrust::device_vector<uint32_t> d_secondary_reference(primary.size(), 0);
    thrust::device_vector<uint32_t> d_secondary = secondary;

    // uint32_t secondary_size = max element in secondary
    uint32_t secondary_size = secondary.size();
    thrust::device_vector<uint32_t> transaction_hits(transactions_count, 1);

    std::vector<std::pair<thrust::host_vector<uint32_t>, thrust::host_vector<uint32_t>>> original_patterns;

    while (number_of_candidates)
    {
        std::cout << "Number of candidates: " << number_of_candidates << std::endl;

        thrust::device_vector<uint32_t> d_candidate_utility(number_of_candidates, 0);
        thrust::device_vector<uint32_t> d_candidate_subtree_utility(number_of_candidates * secondary_size, 0);
        thrust::device_vector<uint32_t> d_candidate_local_utility(number_of_candidates * secondary_size, 0);

        // block size is 32 but grid is number of transactions
        block = dim3(block_size);
        grid = dim3(transactions_count);

        searchGPU<<<grid, block, shared_memory_requirement>>>(d_db, thrust::raw_pointer_cast(transaction_hits.data()), transactions_count,
                                    thrust::raw_pointer_cast(d_candidates.data()), number_of_candidates, candidate_size,
                                    thrust::raw_pointer_cast(d_secondary.data()), secondary_size,
                                    thrust::raw_pointer_cast(d_secondary_reference.data()),
                                    thrust::raw_pointer_cast(d_candidate_utility.data()),
                                    thrust::raw_pointer_cast(d_candidate_subtree_utility.data()),
                                    thrust::raw_pointer_cast(d_candidate_local_utility.data()));

        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipPeekAtLastError());

        thrust::host_vector<uint32_t> h_candidates = d_candidates;
        thrust::host_vector<uint32_t> h_candidate_utility = d_candidate_utility;

        original_patterns.push_back({h_candidates, h_candidate_utility});

        candidate_size += 1;
        thrust::device_vector<uint32_t> d_number_of_new_candidates_per_candidate(number_of_candidates + 1, 0);

        block = dim3(block_size, 1, 1);
        grid = dim3(number_of_candidates / block_size + 1, 1, 1);

        clean_subtree_local_utility<<<grid,block>>>(number_of_candidates, thrust::raw_pointer_cast(d_number_of_new_candidates_per_candidate.data()), 
                                            thrust::raw_pointer_cast(d_candidate_subtree_utility.data()), thrust::raw_pointer_cast(d_candidate_local_utility.data()), 
                                            secondary_size, p.min_utility);

        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipPeekAtLastError());

        uint32_t number_of_new_candidates = thrust::reduce(d_number_of_new_candidates_per_candidate.begin(), d_number_of_new_candidates_per_candidate.end());
        thrust::inclusive_scan(d_number_of_new_candidates_per_candidate.begin(), d_number_of_new_candidates_per_candidate.end(), d_number_of_new_candidates_per_candidate.begin());

        if (number_of_new_candidates == 0)
        {
            break;
        }

        thrust::device_vector<uint32_t> d_new_candidates(number_of_new_candidates * candidate_size, 0);
        thrust::device_vector<uint32_t> d_new_secondary_reference(number_of_new_candidates, 0);

        create_new_candidates<<<grid, block>>>(thrust::raw_pointer_cast(d_candidates.data()), thrust::raw_pointer_cast(d_candidate_subtree_utility.data()), 
                                        number_of_candidates,thrust::raw_pointer_cast(d_new_candidates.data()), 
                                        thrust::raw_pointer_cast(d_new_secondary_reference.data()), secondary_size, candidate_size, 
                                        thrust::raw_pointer_cast(d_number_of_new_candidates_per_candidate.data()));


        number_of_candidates = number_of_new_candidates;
        d_candidates = d_new_candidates;
        d_secondary_reference = d_new_secondary_reference;
        d_secondary = d_candidate_local_utility;

    }

    uint32_t pattern_counter = 0;

    std::cout << "Largest Pattern: " << original_patterns.size() << std::endl;  
    for (uint32_t i = 0; i < original_patterns.size(); i++)
    {
        thrust::host_vector<uint32_t> h_candidates = original_patterns[i].first;
        thrust::host_vector<uint32_t> h_candidate_utility = original_patterns[i].second;

        uint32_t size = i + 1;


        for (uint32_t j = 0; j < h_candidate_utility.size(); j++)
        {
            if (h_candidate_utility[j] < p.min_utility)
            {
                continue;
            }
            for (uint32_t k = 0; k < i + 1; k++)
            {
                // std::cout << h_candidates[j * size + k] << " ";
                // std::cout << intToStr[h_candidates[j * size + k]] << " ";
            }
            // std::cout << "#UTIL: " << h_candidate_utility[j] << std::endl;
            pattern_counter++;
        }
    }
    std::cout << "Number of patterns: " << pattern_counter << std::endl;

}